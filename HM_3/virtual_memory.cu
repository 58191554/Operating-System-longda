#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void init_invert_page_table(VirtualMemory *vm) {
	
  	for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
		// the frame index is i
  	  	vm->invert_page_table[i] = -1;		
  	}
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  	// init variables
  	vm->buffer = buffer;
  	vm->storage = storage;
  	vm->invert_page_table = invert_page_table;
  	vm->pagefault_num_ptr = pagefault_num_ptr;
	
  	// init constants
  	vm->PAGESIZE = PAGESIZE;
  	vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  	vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  	vm->STORAGE_SIZE = STORAGE_SIZE;
  	vm->PAGE_ENTRIES = PAGE_ENTRIES;
	printf("vm->PAGESIZE = %d\n", PAGESIZE);

  	// before first vm_write or vm_read
  	init_invert_page_table(vm);

	// for(int i = 0; i < 100; i++){
	// 	printf("%d ", vm->invert_page_table[i + vm->PAGE_ENTRIES]);
	// }
	
	// init LRU queue
	struct memory_item *tail;
	tail = (struct memory_item*)malloc(sizeof(memory_item));
	tail->up = NULL;
	tail->page_number = -1;			// 0 means no page 
	vm->LRU_bottom = tail;
	vm->LRU_top = tail;

	// initialize the physical memory count as 0
	vm->phyMem_cnt = 0;
}

__device__ void showLRU(VirtualMemory *vm){
	struct memory_item *temp = vm->LRU_bottom;
	while(temp->up != NULL){
		printf("%d ->", temp->page_number);
		temp = temp->up;
	}
	printf("%d\n", temp->page_number);
}

__device__ void update_LRU(VirtualMemory *vm, int page_num){		// put the page_num to the top of the stack

	struct memory_item *temp = vm->LRU_bottom;
	struct memory_item *target = (struct memory_item*)malloc(sizeof(memory_item));

	if(vm->LRU_bottom->page_number == -1){							// initialize the LRU_bottom
		vm->LRU_bottom->page_number = page_num;
		vm->LRU_top->page_number = page_num;
	}
	else if(vm->LRU_top->page_number == page_num){					// already top
		return;	
	}
	else if(vm->LRU_bottom->page_number == page_num){				// remove bottom to top
		vm->LRU_bottom = vm->LRU_bottom->up;
		vm->LRU_top = temp;
		while(temp->up != NULL){
			temp = temp->up;
		}
		target->page_number = page_num;
		temp->up = target;
		// showLRU(vm);
	}
	else{															// find the target and remove to top
		while(temp->up != NULL){
			if(temp->up->page_number == page_num){
				target = temp->up;
				temp->up = target->up;
				temp = temp->up;
				target->up = NULL;
				while(temp->up !=NULL){
					temp = temp->up;
				}
				temp->up = target;
				vm->LRU_top = target;
				// showLRU(vm);
				return;
			}
			temp = temp->up;
		} 
		target->page_number = page_num;
		target->up = NULL;
		temp->up = target;
		vm->LRU_top = target;
		// showLRU(vm);
	}
}

__device__ int get_frameIdx(VirtualMemory *vm, int page_num){
	// find frame index of the given page number in the page table
	for(int i = 0; i < vm->PAGE_ENTRIES; i++){
		if(vm->invert_page_table[i] == page_num){
			return i;
		}
		else if(vm->invert_page_table[i] == -1){		// the page entry is not used
			return i;
		}
	}
	return -1;		// -1 if the page number isn't exist in the page table
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  	/* Complate vm_read function to read single element from data buffer */
	uchar out;	
	int p = addr/vm->PAGESIZE;
	int d = addr%vm->PAGESIZE;
	int f = get_frameIdx(vm, p);

	if(f == -1){
		(*(vm->pagefault_num_ptr))++;
		int victim_p = vm->LRU_bottom->page_number;
		vm->LRU_bottom = vm->LRU_bottom->up;
		int victim_f = get_frameIdx(vm, victim_p);

		for(int i = 0;i<vm->PAGESIZE; i++){
			vm->storage[victim_p*vm->PAGESIZE+i] = vm->buffer[victim_f*vm->PAGESIZE + i];
		}
		f = victim_f;
		vm->invert_page_table[victim_f] = p;
		for(int i = 0; i < vm->PAGESIZE; i++){
			vm->buffer[f*vm->PAGESIZE+i] = vm->storage[p*vm->PAGESIZE + i];
		}
	}
	out = vm->buffer[f*vm->PAGESIZE + d];
	vm->invert_page_table[f] = p;
	if(vm->LRU_top->page_number != p){
		update_LRU(vm, p);
	}

	printf("addr = %d from physical addr = {%d, f = %d}read out = %d\n",addr, f*vm->PAGESIZE + d, f, out);
  	return out; 
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  	/* Complete vm_write function to write value into data buffer */
	int p = addr / vm->PAGESIZE;
	int d = addr % vm->PAGESIZE;
	int f = get_frameIdx(vm, p);				// find the corresponding frame

	if(f == -1){								// if the page number is not in the page table
		(*(vm->pagefault_num_ptr))++;
		int victim_p = vm->LRU_bottom->page_number;
		vm->LRU_bottom = vm->LRU_bottom->up;
		int victim_f = get_frameIdx(vm, victim_p);

		// store victime buffer to disk
		for(int i= 0; i < vm->PAGESIZE; i++){
			vm->storage[victim_p * vm->PAGESIZE + i] = vm->buffer[victim_f*vm->PAGESIZE + i];
		}
		f = victim_f;
		vm->invert_page_table[victim_f] = p;
		// swap target disk to buffer
		for(int i = 0; i < vm->PAGESIZE; i++){
			vm->buffer[f*vm->PAGESIZE + i] = vm->storage[p*vm->PAGESIZE + i];
			// }
		}
	}

	vm->buffer[f*vm->PAGESIZE + d] = value;
	vm->invert_page_table[f] = p;
	if(vm->LRU_top->page_number != p){
		update_LRU(vm, p);
	}

	printf("write buffer: logical address = {%d, p=%d} => physical address = {%d,f=%d} <- %d = %d\n ", 
				addr,p, f*vm->PAGESIZE+d, f, vm->buffer[f*vm->PAGESIZE+d], value);
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset, int input_size) {
  	/* Complete snapshot function togther with vm_read to load elements from data
  	 * to result buffer */
	for(int i = 0; i<input_size; i++){
		int value = vm_read(vm, i);
		results[i + offset] = value;
	}
}
